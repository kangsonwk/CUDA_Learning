
#include <hip/hip_runtime.h>
#include<stdio.h>
// device code
// TAG:
// 1. 分配GPU内存
// 2. 从CPU内存中拷贝数据到GPU中
// 3. 调用CUDA中的内核函数来完成指定的运算
// 4. 将数据从GPU拷回到CPU
// 5. 释放GPU内存空间
__global__ void hello_world(void){
  printf("Hello My first GPU code");
}
int main(int argc,char **argv)
{
  printf("CPU,hello world");
  hello_world<<<1,10>>>();
  // if no this line,it can not output xxx from gpu
  hipDeviceReset();
  return 0;
}