#include <hip/hip_runtime.h>
#include <stdio.h>
// TAG:grid、block、thread之间的关系见1.png

__global__ void checkIndex(int thread_x)
{
  if (threadIdx.x == thread_x)
  {
    printf("threadIdx:(%d,%d,%d),blockIdx:(%d,%d,%d),blockDim:(%d,%d,%d),gridDim(%d,%d,%d)\n",
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z,
           blockDim.x, blockDim.y, blockDim.z,
           gridDim.x, gridDim.y, gridDim.z);
  }
  // 线程索引、块索引、块维度和网格维度
}
int main(int argc, char **argv)
{
  int nElem = 24;
  dim3 block(3, 2, 2);
  dim3 grid(2, (nElem + block.x * block.y - 1) / (block.x * block.y));
  printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
  printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);
  checkIndex<<<grid, block>>>(2);
  hipDeviceReset();
  return 0;
}
